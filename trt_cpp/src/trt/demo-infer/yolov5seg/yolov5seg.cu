#include "hip/hip_runtime.h"
#include <infer/trt-infer.hpp>
#include <common/basic_tools.hpp>
#include <common/cuda-tools.hpp>
#include <common/trt-tensor.hpp>
#include <common/matrix.hpp>
#include <opencv2/opencv.hpp>
#include <demo-infer/yolov5seg/yolov5seg.h>
        // cv::imwrite(to_string(i) + "_.jpg", b);ov5seg.h>

static const int INPUT_H = 640;
static const int INPUT_W = 640;
static const int segWidth = 160;
static const int segHeight = 160;
static const int segChannels = 32;
static const int CLASSES = 80;
static const int Box_col = 117;
static const int Num_box = 25200;
static const int OUTPUT_SIZE = Num_box * (CLASSES+5 + segChannels);  // det output
static const int OUTPUT_SIZE1 = segChannels * segWidth * segHeight ;//seg output

static const float CONF_THRESHOLD = 0.1;
static const float NMS_THRESHOLD = 0.5;
static const float MASK_THRESHOLD = 0.5;


using namespace std;
using namespace cv;

Matrix mygemm(const Matrix& a, const Matrix& b){

    Matrix c(a.rows(), b.cols());
    for(int i = 0; i < c.rows(); ++i){
        for(int j = 0; j < c.cols(); ++j){
            float summary = 0;
            for(int k = 0; k < a.cols(); ++k)
                summary += a(i, k) * b(k, j);

            c(i, j) = summary;
        }
    }
    return c;
}


struct detBox{
    float left, top, right, bottom, confidence;
    int class_label;
    Rect box;
    Mat boxMask;
    Matrix mask_cofs;
    detBox() = default;
    detBox(float left, float top, float right, float bottom, float confidence, int class_label, Matrix mask_cofs, Rect box)
        : left(left), top(top), right(right), bottom(bottom), confidence(confidence), class_label(class_label), mask_cofs(mask_cofs), box(box){}
};

void DrawPred(Mat& img,std::vector<detBox> result) {
	std::vector<Scalar> color;
	srand(time(0));
    for (int i = 0; i < CLASSES; i++)
    {
        int b = rand() % 256;
		int g = rand() % 256;
		int r = rand() % 256;
		color.push_back(Scalar(b, g, r));
    }
    Mat mask = img.clone();
    for (int i = 0; i < result.size(); i++) {
		int left, top;
		left = result[i].box.x;
		top = result[i].box.y;
		int color_num = i;
		rectangle(img, result[i].box,color[result[i].class_label], 2, 8);
        cv::Mat c = mask(result[i].box);

        cv::Mat a = result[i].boxMask;

        c.setTo(color[result[i].class_label], a);
        std::string label = std::to_string(result[i].class_label) + ":" + std::to_string(result[i].confidence);
        int baseLine;
		Size labelSize = getTextSize(label, FONT_HERSHEY_SIMPLEX, 0.5, 1, &baseLine);
		top = max(top, labelSize.height);
		putText(img, label, Point(left, top), FONT_HERSHEY_SIMPLEX, 1, color[result[i].class_label], 2);
	}
	addWeighted(img, 0.5, mask, 0.5, 0, img);

};



void Yolov5Seg::yolov5Seg_inference(){
    // 加载模型
    auto engine = TRT::load_infer("/home/rex/Desktop/tensorrt_learning/trt_cpp/workspace/yolov5s-seg.trtmodel");
    if(!engine){
        printf("load engine failed \n");
        return;
    }
    auto input       = engine->input();
    auto seg_out      = engine->output();
    auto det_out      = engine->output(1);

    int input_width  = input->width();
    int input_height = input->height();
    auto image = imread("/home/rex/Desktop/tensorrt_learning/trt_cpp/workspace/bus.jpg");
    auto img_o = image.clone();
    int img_w = image.cols;
    int img_h = image.rows;
    Mat input_image;
    resize(image,input_image,Size(input_width,input_height));
    Mat show_img = input_image.clone();
    input_image.convertTo(input_image, CV_32F);
    // 预处理
    Mat channel_based[3];
    for(int i = 0; i < 3; ++i)
        channel_based[i] = Mat(input_height, input_width, CV_32F, input->cpu<float>(0, 2-i));

    split(input_image, channel_based);
    for(int i = 0; i < 3; ++i)
        channel_based[i] = (channel_based[i] / 255.0f);
    
    engine->forward(true);

    // 检测结果
    float *det_res = det_out->cpu<float>();
    vector<detBox> boxes;
    for(int i = 0; i < Num_box; ++i){
        float* pitem = det_res + i * Box_col;
        float objness = pitem[4];
        if(objness < CONF_THRESHOLD)
            continue;

        float* pclass = pitem + 5;
        int label     = std::max_element(pclass, pclass + CLASSES) - pclass;

        float prob    = pclass[label];
        float confidence = prob * objness;
        if(confidence < CONF_THRESHOLD)
            continue;

        float cx     = pitem[0];
        float cy     = pitem[1];
        float width  = pitem[2];
        float height = pitem[3];

        float left   = (cx - width * 0.5);
        float top    = (cy - height * 0.5);
        float right  = (cx + width * 0.5);
        float bottom = (cy + height * 0.5);
        Rect rect(left,top,width,height);
        // 每个box的mask系数
        vector<float> temp_proto(pitem + 5 + CLASSES, pitem + 5 + CLASSES + segChannels);
        Matrix tmp_cof(1, segChannels, temp_proto);

        boxes.emplace_back(left, top, right, bottom, confidence, (float)label,tmp_cof,rect);
        
    }

    // NMS
    std::sort(boxes.begin(), boxes.end(), [](detBox &a, detBox &b)
              { return a.confidence > b.confidence; });
    std::vector<bool> remove_flags(boxes.size());
    std::vector<detBox> box_result;
    box_result.reserve(boxes.size());

    auto iou = [](const detBox& a, const detBox& b){
        float cross_left   = std::max(a.left, b.left);
        float cross_top    = std::max(a.top, b.top);
        float cross_right  = std::min(a.right, b.right);
        float cross_bottom = std::min(a.bottom, b.bottom);

        float cross_area = std::max(0.0f, cross_right - cross_left) * std::max(0.0f, cross_bottom - cross_top);
        float union_area = std::max(0.0f, a.right - a.left) * std::max(0.0f, a.bottom - a.top) 
                        + std::max(0.0f, b.right - b.left) * std::max(0.0f, b.bottom - b.top) - cross_area;
        if(cross_area == 0 || union_area == 0) return 0.0f;
        return cross_area / union_area;
    };

    for(int i = 0; i < boxes.size(); ++i){
        if(remove_flags[i]) continue;

        auto& ibox = boxes[i];
        box_result.emplace_back(ibox);
        for(int j = i + 1; j < boxes.size(); ++j){
            if(remove_flags[j]) continue;

            auto& jbox = boxes[j];
            if(ibox.class_label == jbox.class_label){
                // class matched
                if(iou(ibox, jbox) >= NMS_THRESHOLD)
                    remove_flags[j] = true;
            }
        }
    }
    
    // 原型mask 32 * 160 * 160
    float *seg_det = seg_out->cpu<float>();
    vector<float> mask(seg_det, seg_det + segChannels * segWidth * segHeight);
    for (int i = 0; i < mask.size();i++){
        cout << mask[i] << endl;
    }
    // 矩阵表示
    Matrix seg_proto(segChannels, segWidth * segHeight, mask);
    for (int i = 0; i < box_result.size(); ++i) {
        // 可以将所有的mask系数放在一起，然后利用cuda或者其他库进行加速计算
        // 每个目标框的mask系数 乘以原型mask 并取sigmod
        Matrix resSeg = (mygemm(box_result[i].mask_cofs,seg_proto).exp(-1) + 1.0).power(-1);
        
        Mat resMat(resSeg.data_);
        resMat = resMat.reshape(0,{segHeight,segWidth});
        // 如果图片预处理为直接resize,那么计算出来的resMat可以直接缩放回原图，
        // 如果是填充黑边的resize，可以参考原代码将原型mask恢复到原图大小
        resize(resMat, resMat, Size(INPUT_H,INPUT_W), INTER_NEAREST);
        // 获取原型mask里面目标框的区域
        Rect temp_rect = box_result[i].box;
        // 将目标框区域 大于0.5的值变为255
        cv::Mat binaryMat;
        inRange(resMat(temp_rect), 0.5, 1, binaryMat);
		box_result[i].boxMask = binaryMat;
        // cv::imwrite(to_string(i) + "_.jpg", b);
    }
    // 渲染
    DrawPred(show_img, box_result);
    cv::imwrite("output-seg.jpg", show_img);
}